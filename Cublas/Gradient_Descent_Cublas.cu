#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "mat.h"

#include <hipblas.h>

#include <hip/hip_runtime.h>
// Matrices are stored in column-major order: 
// M(row, col) = M.elements[row + col*M.height]
typedef struct {
    	int width;    
 	int height;    
	double* elements; 
	}Matrix;

//Vector * Matrix
//C(1,M)=A(1,N)*B(N,M)
void Vec_x_Matrix(hipblasHandle_t &handle,double A[],Matrix B,double C[],int N,int M){
	int lda=1,ldb=N,ldc=1;
	const double alf=1;
	const double bet=0;
	const double*alpha = &alf;
	const double*beta=&bet;
		
	//Do the operation
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1,M,N,alpha,A,lda,B.elements,ldb,beta,C,ldc);
}

//Vector Substraction
void Vec_sub(double A[],double B[],double C[],int N){
	int i;
	for(i = 0; i < N; i++){
		C[i] = A[i]-B[i];
	}
}

//Constant x Vector
void Con_x_Vector(double Con, double * Vec, int N,double* Vec_R){
	int i;
	for(i=0;i<N;i++){
		Vec_R[i]=Vec[i]*Con;
	}
}

//Get sub_matrix from matrix
void Sub_Matrix(Matrix m,Matrix Sub_m,int N,int M,int start,int end){
	int i,j;
	for(i=0; i< N; i++){
		for(j=start; j <end+1; j++){
			Sub_m.elements[i+(j-start)*Sub_m.height] = m.elements[i+j*m.height];
		}
	}
}

void Gradient_descent(Matrix A, Matrix Series,int max_iter, double delta,int N,int T){
	double* Aind = (double*)malloc(N*sizeof(double));
	double* Series_sub2ind = (double*)malloc((T-1)*sizeof(double));
	// allocate the space for Series_sub1
	Matrix Series_sub1;
	Series_sub1.width = T-1;
	Series_sub1.height = N;
	size_t size = Series_sub1.width*Series_sub1.height*sizeof(double);
	Series_sub1.elements = (double*)malloc(size);

	// allocate the space for Series_sub2
	Matrix Series_sub2;
	Series_sub2.width = T-1;
	Series_sub2.height = N;
	size = Series_sub2.width*Series_sub2.height*sizeof(double);
	Series_sub2.elements = (double*)malloc(size);

	
	// allocate the space for Series_sub1_Transpose
	Matrix Series_sub1_Transpose;
	Series_sub1_Transpose.width = N;
	Series_sub1_Transpose.height = T-1;
	size = Series_sub1_Transpose.width*Series_sub1_Transpose.height*sizeof(double);
	Series_sub1_Transpose.elements = (double*)malloc(size);
	
	// compute sub matrix
	Sub_Matrix(Series,Series_sub1,N,T,0,T-2);
	Sub_Matrix(Series,Series_sub2,N,T,1,T-1);
	// transpose sub matrix1
	int i,j,ind;
	for(i = 0;i<T-1;i++){
		for(j=0;j<N;j++){
			Series_sub1_Transpose.elements[i+j*Series_sub1_Transpose.height]=Series_sub1.elements[j+i*Series_sub1.width];
		}
	}
	for(ind =0;ind++;ind<N){
		for(i =0; i < N; i++){
			Aind[i]=A.elements[ind+i*A.height];
		}
		for(i=0; i < T-1; i++){
			Series_sub2ind[i]=Series_sub2.elements[i*Series_sub2.height+ind];
		}
	}

	//allocate the device memory for Series_sub1,Series_sub1_Transpose,Temp,A,Series_sub2
	Matrix d_Series_sub1;
	d_Series_sub1.width = T-1;
	d_Series_sub1.height = N;
	size = d_Series_sub1.width*d_Series_sub1.height*sizeof(double);
	hipMalloc(&d_Series_sub1.elements,size);
	hipMemcpy(d_Series_sub1.elements,Series_sub1.elements,size,hipMemcpyHostToDevice);

	Matrix d_Series_sub2;
	d_Series_sub2.width = T-1;
	d_Series_sub2.height = N;
	size = d_Series_sub2.width*d_Series_sub2.height*sizeof(double);
	hipMalloc(&d_Series_sub2.elements,size);
	hipMemcpy(d_Series_sub2.elements,Series_sub2.elements,size,hipMemcpyHostToDevice);

	Matrix d_Series_sub1_Transpose;
	d_Series_sub1_Transpose.width = N;
	d_Series_sub1_Transpose.height = T-1;
	size = d_Series_sub1_Transpose.width*d_Series_sub1_Transpose.height*sizeof(double);
	hipMalloc(&d_Series_sub1_Transpose.elements,size);
	hipMemcpy(d_Series_sub1_Transpose.elements,Series_sub1_Transpose.elements,size,hipMemcpyHostToDevice);

	Matrix d_A;
	d_A.width =A.width;
	d_A.height = A.height;
	size = d_A.width*d_A.height*sizeof(double);
	hipMalloc(&d_A.elements,size);
	hipMemcpy(d_A.elements,A.elements,size,hipMemcpyHostToDevice);
	//copy Aind and Series_sub2ind to device memory
	double* d_Aind;
	hipMalloc(&d_Aind,N*sizeof(double));
	hipMemcpy(d_Aind,Aind,N*sizeof(double),hipMemcpyHostToDevice);
	
	double* d_Series_sub2ind;
	hipMalloc(&d_Series_sub2ind,(T-1)*sizeof(double));
	hipMemcpy(d_Series_sub2ind,Series_sub2ind,(T-1)*sizeof(double),hipMemcpyHostToDevice);
	
	double* d_Temp; 
	hipMalloc(&d_Temp,(T-1)*sizeof(double));

	double* d_G;
	hipMalloc(&d_G,N*sizeof(double)); 
	
	//create handle for Cublas
	hipblasHandle_t handle;
	hipblasCreate(&handle);	


	for(ind=0; ind<N; ind++){	
		for (i = 0; i < max_iter; i++){
			// compute G
			Vec_x_Matrix(handle,d_Aind,d_Series_sub1,d_Temp,N,T-1);
			double alf =-1;
			const  double *alpha = &alf;
			hipblasDaxpy(handle,T-1,alpha,d_Series_sub2ind,1,d_Temp,1);
			Vec_x_Matrix(handle,d_Temp,d_Series_sub1_Transpose,d_G,T-1,N);
			// compute the A[ind] in the next iteratio
			double alf_1=-delta;
			const double *alpha_1 = &alf_1;
			hipblasDaxpy(handle,N,alpha_1,d_G,1,d_Aind,1);
		}
	}

	hipblasDestroy(handle);
	// Free the tempory memomry
	free(Series_sub1_Transpose.elements);
	free(Series_sub2.elements);
	free(Series_sub1.elements);
	free(Series_sub2ind);
	free(Aind);
	hipFree(d_G);
	hipFree(d_Temp);
	hipFree(d_Series_sub2ind);
	hipFree(d_Aind);
	hipFree(d_A.elements);
	hipFree(d_Series_sub1_Transpose.elements);
	hipFree(d_Series_sub2.elements);
	hipFree(d_Series_sub1.elements);
}
	
int main(int argc, char** argv){
	int T = 200;
	int N = 100;
	double delta = (double)pow(10,-5);
	int max_iter = 100;
	size_t size;
	
	//load Series from .mat file
	MATFile *pmat;	
	const char* file ="Series.mat";
	const char* varname="Series";
	mxArray* Series_mat;
	pmat = matOpen(file, "r");
	if(pmat == NULL){
		printf("Error reopening file%s\n", file);
		return(NULL);
	}
	
	Series_mat =  matGetVariable(pmat, varname);
	if(Series_mat == NULL){
		printf("Error reading in file%s\n", file);
		return(NULL);
	}
	
	matClose(pmat);
	
	mwSize row, col; // mwSize is int 
	mwSize nRow = mxGetM(Series_mat); 
	mwSize nCol = mxGetN(Series_mat);
	double *Series_Pr = mxGetPr(Series_mat);
	
	Matrix Series;
	Series.width = 	nCol;
	Series.height = nRow;
	size = Series.width*Series.height*sizeof(double);
	Series.elements = (double*)malloc(size);
	for(row = 0; row < nRow; row++) {
		for(col = 0; col < nCol; col++) {
			Series.elements[row+col*Series.height] = Series_Pr[nRow * col + row];// this needs testing
		}
	}
	
	int i,j;
	Matrix A;
	//create matrix A
	A.width = N;
	A.height = N;
	size = A.width*A.height*sizeof(double);
	A.elements = (double*)malloc(size);
	//time counter
	double tstart,tstop,ttime;
	
	tstart =(double)clock()/CLOCKS_PER_SEC;
	int counter = 0;
	while(counter<10){
		//initialize A	
		for(i =0; i < A.height; i++){
			for(j = 0; j < A.width; j++){
				A.elements[i+j*A.height] = 0;
			}
		}
		
		// Gradient_descent
		Gradient_descent(A,Series,max_iter,delta,N,T);
		counter++;
	}
	tstop = (double)clock()/CLOCKS_PER_SEC;
	ttime=tstop-tstart;
	printf("time:%fs\n",ttime);
	free(A.elements);
	free(Series.elements);
	return 0;
}
